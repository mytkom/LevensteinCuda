#include <iostream>
#include <string>
#include <vector>
#include <memory>
#include <algorithm>
#include <chrono>
#include <unistd.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utils.hpp"
#include "cpu_calculator.hpp"
#include "gpu_calculator.hpp"
#include "kernel.h"

std::string alphabet = "ACGT";

int main(int argc, const char** argv) {
  bool isDMatricesIdentical = true, verbose = false, isCuda = false;

  if(argc < 3) {
    std::cout << "Usage:" << std::endl;
    std::cout << "  levcuda [target string filepath] [source string filepath] (-v|-a arg)" << std::endl;
    std::cout << "  -v: verbose" << std::endl;
    std::cout << "  -a [filepath to alphabet]: specify different alphabet" << std::endl;
    return EXIT_FAILURE;
  }

  for(int i = 3; i < argc; ++i)
  {
    if(strcmp(argv[i], "-v") == 0) {
      verbose = true;
    } else if(strcmp(argv[i], "-a") == 0 && argc != i + 1) {
      alphabet = read_file(argv[i+1]);
      i++;
    }
  }

  std::string s1 = read_file(argv[1]);
  std::string s2 = read_file(argv[2]);

  if(verbose) {
    std::cout << "Target string: " << s1 << std::endl;
    std::cout << "Source string: " << s2 << std::endl;
    std::cout << std::endl;
  }

  std::cout << "CPU calculation in progress" << std::endl;
  
  CpuCalculator calc(s1, s2);

  auto tic = std::chrono::high_resolution_clock::now();
  calc.Calculate();
  auto toc = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> cpuTime = toc - tic;

  std::cout << "CPU ms: " << cpuTime.count() << std::endl;

  if(verbose) calc.Print();
  std::vector<std::string> transformationsStrings = calc.GetTransformations();
  save_edits_to_file(transformationsStrings, "./cpu_transformations.txt");

  // Gpu calculation if Gpu available
  if (isCuda = (hipSetDevice(0) == hipSuccess)) {
    std::cout << std::endl << "GPU calculation in progress" << std::endl;
    GpuCalculator gpuCalc(s1, s2, alphabet);

    float gpuTime;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    gpuCalc.Calculate();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    std::cout << "GPU ms: " << gpuTime << std::endl;
    std::cout << "CPU/GPU ratio: " << cpuTime.count()/gpuTime << std::endl;

    if(verbose) gpuCalc.Print();
    transformationsStrings = gpuCalc.GetTransformations();
    save_edits_to_file(transformationsStrings, "./gpu_transformations.txt");

    for(int i = 0; i <= s2.size(); ++i) {
      for(int j = 0; j <= s1.size(); ++j) {
        if(gpuCalc.dMatrix[i*(s1.size()+1) + j] != calc.dMatrix[i][j])
          isDMatricesIdentical= false;
      }
    }
  } else {
    std::cout << "Cuda device not found!" << std::endl;
  }

  std::cout << std::endl << "Distance: " << calc.dMatrix[s2.size()][s1.size()] << std::endl;

  if(isCuda)
    std::cout << "Identical distance matrices?: " << (isDMatricesIdentical ? "true" : "false") << std::endl;

  return EXIT_SUCCESS;
}
